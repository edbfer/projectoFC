#include "hip/hip_runtime.h"
#include "cuda.cuh"

__device__ float cG;
__device__ float cgama;
__device__ float comega;
__device__ float cdt;
__device__ float ch;
extern float h;

void cuda_setup(float G, float gama, float omega, float dt, float h)
{
  hipMemcpyToSymbol(HIP_SYMBOL("cG"), &cG, sizeof(float), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL("cgama"), &cgama, sizeof(float), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL("comega"), &comega, sizeof(float), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL("cdt"), &cdt, sizeof(float), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL("ch"), &ch, sizeof(float), 0, hipMemcpyHostToDevice);
}

float cuda_norm(matriz& l)
{
  complex* val;
  float* linhas;
  hipMalloc(&val, sizeof(complex)*l.n*l.m);
  hipMalloc(&linhas, sizeof(float)*l.n);
  hipDeviceSynchronize();

  hipMemcpy(val, l.mat, sizeof(complex)*l.n*l.m, hipMemcpyHostToDevice);

  dim3 tpb(8);
  dim3 nb(l.n / tpb.x);
  cuda_reduction<<<nb, tpb>>>(l.n, l.m, val, linhas);

  float* host = new float[l.n];
  hipMemcpy(host, linhas, sizeof(float)*l.n, hipMemcpyDeviceToHost);

  float v = 0;
  for(int i = 0; i<l.n; i++)
  {
    v = v + host[i];
  }

  hipFree(val);
  hipFree(linhas);
  delete[] host;

  return v*h*h;

}

__global__ void cuda_reduction(int n, int m, complex* val, float* linhas)
{
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;

  if((x > n) || (x < 0))
    return;

  float v = 0;
  for(int i = 0; i < m; i++)
  {
    float mod = val[x * m + i].mod();
    v = v + mod*mod;
  }

  linhas[x] = v;
}

matriz cuda_doround(matriz& l)
{
  complex* psi1, *lmat, *psi2, *r;

  int n = l.n, m = l.m;
  hipMalloc(&lmat, sizeof(complex)*n*m);
  hipMalloc(&psi1, sizeof(complex)*n*m);
  hipMalloc(&psi2, sizeof(complex)*n*m);
  hipMalloc(&r, sizeof(complex)*n*m);
  hipDeviceSynchronize();

  hipMemcpy(lmat, l.mat, sizeof(complex)*n*m, hipMemcpyHostToDevice);

  dim3 tpb(24, 24);
  dim3 nb(n / tpb.x, m / tpb.y);
  cuda_psi1<<<nb, tpb>>>(lmat, psi1, n, m);
  hipDeviceSynchronize();
  cuda_psi2<<<nb, tpb>>>(lmat, psi1, psi2, n, m);
  hipDeviceSynchronize();
  cuda_psin<<<nb, tpb>>>(lmat, psi2, r, n, m);
  hipDeviceSynchronize();

  matriz res(128, 128);

  hipMemcpy(res.mat, r, sizeof(complex)*n*m, hipMemcpyDeviceToHost);

  float norm = cuda_norm(res);
  res = res * complex(1/norm, 0.0f);

  hipFree(lmat);
  hipFree(psi1);
  hipFree(psi2);
  hipFree(r);
  return res;

}

__device__ complex cuda_f(int i, int j, complex c, complex c1, complex c2, complex c3, complex c4)
{
  float x = -10.0f + (i-1)*ch;
  float y = -10.0f + (j-1)*ch;

  complex lapx = (c2 - 2.0f*c + c1)/(ch*ch);
  complex lapy = (c4 - 2.0f*c + c3)/(ch*ch);
  complex lap = (lapx + lapy)*(-0.5f);

  complex p2 = c * ((x*x) + (y*y))/(2.0f);
  float n = c.mod();
  n *= n;
  complex p3 = c * (cG*n);

  complex dx = (c2 - c1)/(2.0f*ch);
  complex dy = (c4 - c3)/(2.0f*ch);
  dx = dx * y;
  dy = dy * x;
  complex p4 = (dy - dx) * complex(0, comega);

  complex r = lap + p2 + p3 - p4;
  complex m = complex(-cgama, 1.0f);
  complex res = r/m;
  return res;
}

__global__ void cuda_psi1(complex* l, complex* r, int n, int m)
{
  //RungeKuttadevMat
  int i = (blockIdx.x * blockDim.x) + threadIdx.x;
  int j = (blockIdx.y * blockDim.y) + threadIdx.y;

  if((i <= 0) || (i >= 127) || (j <= 0) || (j >= 127))
    return;

  complex c = l[i * m + j];
  complex c1 = l[(i-1) * m + j];
  complex c2 = l[(i+1) * m + j];
  complex c3 = l[i * m + (j-1)];
  complex c4 = l[i * m + (j+1)];

  complex ft = cuda_f(i, j, c, c1, c2, c3, c4);
  r[i * m + j] = c + (ft*cdt);
}

__global__ void cuda_psi2(complex* l, complex* psi1, complex* r, int n, int m)
{
  //RungeKutta
  int i = (blockIdx.x * blockDim.x) + threadIdx.x;
  int j = (blockIdx.y * blockDim.y) + threadIdx.y;

  if((i <= 0) || (i >= 127) || (j <= 0) || (j >= 127))
    return;

  complex c = psi1[i * m + j];
  complex la = l[i * m + j];
  complex c1 = psi1[(i-1) * m + j];
  complex c2 = psi1[(i+1) * m + j];
  complex c3 = psi1[i * m + (j-1)];
  complex c4 = psi1[i * m + (j+1)];

  complex ft = cuda_f(i, j, c, c1, c2, c3, c4);
  ft = ft*0.25f*cdt;
  complex ant = c*0.25f;
  complex lat = la * 0.75f;

  r[i * m + j] = ft + ant + lat;
}

__global__ void cuda_psin(complex* l, complex* psi2, complex* r, int n, int m)
{
  //RungeKutta
  int i = (blockIdx.x * blockDim.x) + threadIdx.x;
  int j = (blockIdx.y * blockDim.y) + threadIdx.y;

  if((i <= 0) || (i >= 127) || (j <= 0) || (j >= 127))
    return;

  complex c = psi2[i * m + j];
  complex la = l[i * m + j];
  complex c1 = psi2[(i-1) * m + j];
  complex c2 = psi2[(i+1) * m + j];
  complex c3 = psi2[i * m + (j-1)];
  complex c4 = psi2[i * m + (j+1)];

  complex ft = cuda_f(i, j, c, c1, c2, c3, c4);
  ft = ft * (cdt * (2.0f/3.0f));
  complex ant = c * (2.0f/3.0f);
  complex lat = la * (1.0f/3.0f);

  r[i * m + j] = ft + ant + lat;
}
